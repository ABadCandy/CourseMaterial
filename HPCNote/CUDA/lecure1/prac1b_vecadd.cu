#include "hip/hip_runtime.h"
//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>


//
// kernel routine
// modify my_first_kernel function to vector add.
// computes the vector addition of A and B into C. The 3 vectors have the same number of elements numElements.
//

__global__ void my_first_kernel(const float *A, const float *B, float *C, int numElements)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if (tid < numElements)
  {
      C[tid] = A[tid] + B[tid];
  }
}


//
// main code
//

int main(int argc, const char **argv)
{
  float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
  int   nblocks, nthreads, nsize, n; 

  // initialise card

  findCudaDevice(argc, argv);

  // set number of blocks, and threads per block

  nblocks  = 2;
  nthreads = 8;
  nsize    = nblocks*nthreads ;

  // allocate host memory for array

  h_A = (float *)malloc(nsize*sizeof(float));
  h_B = (float *)malloc(nsize*sizeof(float));
  h_C = (float *)malloc(nsize*sizeof(float));
  
  // initialize the host input vectors

  for (int i = 0; i < nsize; ++i)
  {
      h_A[i] = rand()/(float)RAND_MAX;
      h_B[i] = rand()/(float)RAND_MAX;
  }
  
  // allocate device memory for array

  checkCudaErrors(hipMalloc((void **)&d_A, nsize*sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_B, nsize*sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_C, nsize*sizeof(float)));

  // copy the host input vectors A and B in host memory to the device input vectors in
  // device memory

  checkCudaErrors( hipMemcpy(d_A, h_A, nsize*sizeof(float), hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(d_B, h_B, nsize*sizeof(float), hipMemcpyHostToDevice) );
  
  // execute kernel
  
  my_first_kernel<<<nblocks,nthreads>>>(d_A, d_B, d_C, nsize);
  getLastCudaError("my_first_kernel execution failed\n");

  // copy back results and print them out

  checkCudaErrors( hipMemcpy(h_C, d_C, nsize*sizeof(float), hipMemcpyDeviceToHost) );

  for (n=0; n<nsize; n++) 
  {
      printf(" %f + %f = %f \n",h_A[n],h_B[n],h_C[n]);
	  if (fabs(h_A[n] + h_B[n] - h_C[n]) > 1e-5)
      {
          fprintf(stderr, "Result verification failed at element %d!\n", n);
          exit(EXIT_FAILURE);
      }
  }
  // free memory 

  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));
  free(h_A);
  free(h_B);
  free(h_C);

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

  return 0;
}
