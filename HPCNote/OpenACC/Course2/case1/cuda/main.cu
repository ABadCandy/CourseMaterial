
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define N (1<<24)
#define blocksize 1024
#define blocknumb (N/blocksize)

#define checkCudaAPIErrors(F) if ((F) != hipSuccess) \
{ printf("Error at line %d in file %s: %s\n", __LINE__, __FILE__, hipGetErrorString(hipGetLastError())); exit(-1); }

__global__ void vecDot(double *a, double *b, double *sub_sum)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ double component[blocksize];

    component[threadIdx.x] = a[gid] * b[gid];

    __syncthreads();
    for (int i=(blocksize>>1); i>0; i=(i>>1))
    {
        if (threadIdx.x < i)
            component[threadIdx.x] += component[threadIdx.x + i];
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        sub_sum[blockIdx.x] = component[0];
    }
}


int main()
{
    int i, device = 0;
    double *h_a, *h_b, *h_c;
    double *d_a, *d_b, *d_c;
    double *h_subSum;
    double *d_subSum;

    struct timeval start;
    struct timeval end;
    double elapsedTime;
    double sum_cpu = 0.0;
    double sum_gpu = 0.0;
    hipDeviceProp_t prop;

    h_a = (double *)malloc(sizeof(double) * N);
    h_b = (double *)malloc(sizeof(double) * N);
    h_c = (double *)malloc(sizeof(double) * N);
    h_subSum = (double *)malloc(sizeof(double) * blocknumb);


    // init a and b
    for (i=0; i<N; i++)
    {
        h_a[i] = (double)rand()/RAND_MAX;
        h_b[i] = (double)rand()/RAND_MAX;
        h_c[i] = h_a[i] * h_b[i];

        sum_cpu += h_c[i];
    } 

    hipSetDevice(device);
    hipGetDeviceProperties(&prop, device);
    printf("Using gpu %d: %s\n", device, prop.name);

    // timer begin
    gettimeofday(&start, NULL);

    hipMalloc((void**)&d_a, sizeof(double) * N);
    hipMalloc((void**)&d_b, sizeof(double) * N);
    hipMalloc((void**)&d_c, sizeof(double) * N);
    hipMalloc((void**)&d_subSum, sizeof(double) * blocknumb);

    checkCudaAPIErrors(hipMemcpy(d_a, h_a, sizeof(double) * N, hipMemcpyHostToDevice));
    checkCudaAPIErrors(hipMemcpy(d_b, h_b, sizeof(double) * N, hipMemcpyHostToDevice));

    vecDot<<<blocknumb, blocksize>>>(d_a, d_b, d_subSum);
    checkCudaAPIErrors(hipMemcpy(h_subSum, d_subSum, sizeof(double) * blocknumb, hipMemcpyDeviceToHost));

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_subSum);
    
    for (i=0; i<blocknumb; i++)
    {
        sum_gpu += h_subSum[i];
    }
    // timer end 
    gettimeofday(&end, NULL);

    elapsedTime  = (end.tv_sec - start.tv_sec) * 1000.0;    // sec to ms
    elapsedTime += (end.tv_usec - start.tv_usec) / 1000.0;  // us  to ms

    printf("the result on GPU is %lf\n", sum_gpu);
    printf("the result on CPU is %lf\n", sum_cpu);
    printf("the elapsedTime is %f ms\n", elapsedTime);

    free(h_a);
    free(h_b);
    free(h_c);
    free(h_subSum);

    return 0;
}
